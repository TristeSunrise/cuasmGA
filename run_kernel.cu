#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            const char* errStr; \
            hipDrvGetErrorString(err, &errStr); \
            std::cerr << "CUDA Error: " << errStr << std::endl; \
            exit(1); \
        } \
    } while (0)

int main(int argc, char** argv) {
    if (argc < 2) {
        std::cerr << "Usage: ./run_kernel your_kernel.cubin" << std::endl;
        return 1;
    }

    const char* cubin_file = argv[1];

    // Init
    CHECK_CUDA(hipInit(0));
    hipDevice_t dev;
    CHECK_CUDA(hipDeviceGet(&dev, 0));
    hipCtx_t ctx;
    CHECK_CUDA(hipCtxCreate(&ctx, 0, dev));

    // Load cubin
    hipModule_t mod;
    CHECK_CUDA(hipModuleLoad(&mod, cubin_file));

    // Get function (use your kernel name!)
    hipFunction_t func;
    CHECK_CUDA(hipModuleGetFunction(&func, mod, "mmleakyrelu"));
    //need to align with triton kernel name

    // Allocate input/output
    const int N = 1024;
    hipDeviceptr_t d_in, d_out;
    CHECK_CUDA(hipMalloc(&d_in, N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_out, N * sizeof(float)));

    void* args[] = { &d_out, &d_in, &N };
    //need to align with triton kernel input args

    // Launch
    auto start = std::chrono::high_resolution_clock::now();
    //need to align with triton kernel config
    CHECK_CUDA(hipModuleLaunchKernel(func,
        1, 1, 1,       // grid
        256, 1, 1,     // block
        0, 0, args, 0));
    CHECK_CUDA(hipCtxSynchronize());
    auto end = std::chrono::high_resolution_clock::now();

    double latency_ms = std::chrono::duration<double, std::milli>(end - start).count();
    std::cout << "Latency: " << latency_ms << " ms" << std::endl;

    // Cleanup
    hipFree(d_in);
    hipFree(d_out);
    hipModuleUnload(mod);
    hipCtxDestroy(ctx);

    return 0;
}
